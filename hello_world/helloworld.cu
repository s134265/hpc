#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void printgpu(){
  printf("Hello world! I’m thread %d out of 64 in block %d. My global thread id is %d out of 256.\n", threadIdx.x, blockIdx.x, blockIdx.x*blockDim.x+threadIdx.x);
}


int main(int argc, char *argv[])
{
  printgpu<<<4,64>>>();
}