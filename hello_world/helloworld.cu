#include "hip/hip_runtime.h"
#include <stdio.h>


__global__ void printgpu{
  printf("Hello world! I’m thread %d out of 64 in block %d. My global thread id is %d out of 256.\n", blockIdx.x, blockDim.x, blockIdx.x*blockDim.x)
  return 0;
}


int main()
{
  printgpu<<<4,64>>>;
}