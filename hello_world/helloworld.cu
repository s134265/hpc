
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void printgpu(){
  printf("Hello world! I’m thread %d out of 64 in block %d. My global thread id is %d out of 256.\n", threadIdx.x, blockIdx.x, blockIdx.x*blockDim.x+threadIdx.x);
}


int main()
{
  printgpu<<<4,64>>>();
}