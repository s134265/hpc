#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>

__global__ void printgpu(){
  if (blockIdx.x*blockDim.x+threadIdx.x+1==100) {
  	int *a = (int*) 0x10000; *a = 0;
  }
  	printf("Hello world! I’m thread %d out of %d in block %d. My global thread id is %d out of %d.\n", threadIdx.x, blockDim.x ,blockIdx.x, blockIdx.x*blockDim.x+threadIdx.x+1,gridDim.x*blockDim.x);
}


int main(int argc, char *argv[])
{
  int blockSize = atoi(argv[1]);
  int blocks = atoi(argv[2]);
  printgpu<<<blocks,blockSize>>>();
  hipDeviceSynchronize();
}