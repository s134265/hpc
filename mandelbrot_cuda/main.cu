#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include "mandelgpu.h"


int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    //int   *image, 
    int *h_image, *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    dim3 dimGrid(width/16+1,height/16+1,1);
    dim3 dimBlock(16,16,1);
    
    
    /*
    double start = omp_get_wtime();
    image = (int *)malloc( width * height * sizeof(int));
    #pragma omp parallel
    {
    mandel(width, height, image, max_iter);
    }
    free(image);
    writepng("mandelbrot.png", image, width, height);
    double end = omp_get_wtime() - start;
    printf("Wall time cpu: %lf",end);
    */
    
    h_image = (int *)malloc( width * height * sizeof(int));
    hipMalloc((void**)&d_image,width * height * sizeof(int));
    double start = omp_get_wtime();
    mandelgpu<<<dimGrid,dimBlock>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();
    hipMemcpy(h_image,d_image,width * height * sizeof(int),hipMemcpyDeviceToHost);
    double end = omp_get_wtime() - start;
    writepng("mandelbrot.png", h_image, width, height);
    hipFree(d_image);
    free(h_image);
    
    printf("Wall time gpu: %lf",end);
    //

    return(0);
}
