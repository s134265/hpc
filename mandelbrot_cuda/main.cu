#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include "mandelgpu.h"


int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    //int   *image, 
    int *image, *h_image, *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    dim3 dimGrid(width/16+1,height/16+1,1);
    dim3 dimBlock(16,16,1);
    
    


    
    image = (int *)malloc( width * height * sizeof(int));
    double start = omp_get_wtime();
    #pragma omp parallel
    {
    mandel(width, height, image, max_iter);
    }
    double end = omp_get_wtime() - start;
    writepng("mandelbrot.png", image, width, height);
    free(image);
    
    printf("Wall time cpu: %lf\n",end);
    
    h_image = (int *)malloc( width * height * sizeof(int));
    hipMalloc((void**)&d_image,width * height * sizeof(int));
    start = omp_get_wtime();
    mandelgpu<<<dimGrid,dimBlock>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();
    hipMemcpy(h_image,d_image,width * height * sizeof(int),hipMemcpyDeviceToHost);
    end = omp_get_wtime() - start;
    writepng("mandelbrot.png", h_image, width, height);
    hipFree(d_image);
    free(h_image);
    
    printf("Wall time gpu: %lf\n",end);
    //

    return(0);
}
