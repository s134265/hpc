#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "mandel.h"
#include "writepng.h"
#include "mandelgpu.h"


int
main(int argc, char *argv[]) {

    int   width, height;
    int	  max_iter;
    int   *image, *h_image, *d_image;

    width    = 2601;
    height   = 2601;
    max_iter = 400;

    // command line argument sets the dimensions of the image
    if ( argc == 2 ) width = height = atoi(argv[1]);
    
    
    
    
    double start = omp_get_wtime();
    image = (int *)malloc( width * height * sizeof(int));
    #pragma omp parallel
    {
    mandel(width, height, image, max_iter);
    }
    free(image);
    writepng("mandelbrot.png", image, width, height);
    double end = omp_get_wtime() - start;
    printf("Wall time cpu: %lf",end);

    start = omp_get_wtime();
    h_image = (int *)malloc( width * height * sizeof(int));
    hipMalloc((void**)&d_image,width * height * sizeof(int));
    mandelgpu<<<1,1>>>(width, height, d_image, max_iter);
    hipDeviceSynchronize();
    hipMemcpy(h_image,d_image,width * height * sizeof(int),hipMemcpyDeviceToHost);
    
    writepng("mandelbrot.png", h_image, width, height);
    hipFree(d_image);
    free(h_image);
    end = omp_get_wtime() - start;
    printf("Wall time gpu: %lf",end);
    //

    return(0);
}
