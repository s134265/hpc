
#include <hip/hip_runtime.h>
__global__ void
matmultvec(int m, double *A, double *b, double *c){

    int i, j;
    i = threadIdx.x+blockIdx.x*blockDim.x;
    for (j=0; j<m; j++){
       c[i] += A[i+j] * b[j];
    }
}
