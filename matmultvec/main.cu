#include <stdio.h>
#include <stdlib.h>
#include <omp.h>
#include <hip/hip_runtime_api.h>
#include "datatools.h"
#include "matadd.h"



int
main(int argc, char *argv[]) {

    double *h_A,*h_b,*h_c,*d_A,*d_b,*d_c;
    int i, j, m=32, n=5;
    h_A = (double *)malloc(m*n * sizeof(double));
    h_b = (double *)malloc(n * sizeof(double));
    h_c = (double *)malloc(m * sizeof(double));
    hipMalloc((void **)&d_A,m*n * sizeof(double));
    hipMalloc((void **)&d_b,n * sizeof(double));
    hipMalloc((void **)&d_c,m * sizeof(double));
    init_1d(n,m,h_b,h_c);
    init(m,n,h_A);
    hipMemcpy(d_A, h_A, m*n * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, m * sizeof(double),hipMemcpyHostToDevice);
    matmultvec<<<m/16,16>>>(m, d_A, d_b, d_c);
    hipDeviceSynchronize();
    hipMemcpy(h_c, d_c, m * sizeof(double),hipMemcpyDeviceToHost);
    for (j=0;j<n;j++){    
         printf("%lf ", h_b[j]);
    }
    for (i=0;i<m;i++){
	printf("%lf ", h_c[i]);
    }
    free(h_A);
    free(h_b);
    free(h_c);
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_c);
}
